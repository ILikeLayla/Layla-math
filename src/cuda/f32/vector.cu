
#include <hip/hip_runtime.h>
#include <stdio.h>


void print_vector(float *v, unsigned int lenth) {
    for (int i = 0; i < lenth; i++) {
        printf("%f ", v[i]);
    }
    printf("\n");
}

extern "C" {
    void print_vector_f32_unsafe(float *v, unsigned int lenth) {
        print_vector(v, lenth);
    }
}