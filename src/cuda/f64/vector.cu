
#include <hip/hip_runtime.h>
#include <stdio.h>

void print_vector(double *v, unsigned int lenth) {
    for (int i = 0; i < lenth; i++) {
        printf("%f ", v[i]);
    }
    printf("\n");
}
extern "C" {
    void print_vector_f64_unsafe(double *v, unsigned int lenth) {
        print_vector(v, lenth);
    }
}